#include "hip/hip_runtime.h"
#include "../../../include/examples/integration/integration.hpp"

#include "../../../../src/lattice_model_impl/thrust/thrust_integration.cu"
#include "../../../../src/lattice_model_impl/thrust/thrust_finite_integration.cu"

struct test_func : public thrust::unary_function<double, double>
{
    __host__ __device__
    double operator() (const double x)
    {
        return thrust::exp(thrust::complex<double>{(std::pow(x, 2) * std::sin(x) * std::exp(-1.0) * std::sin(x) * std::exp(-1.0) * std::sin(x) * std::exp(-1.0)), 0.0}).real();
    }
};

template<typename Integrator>
void test_integration(const int n=100)
{
    test_func thrust_func;

    Integrator integrator(n);

    std::pair<double, double> result;
    for(auto i = 0; i < 100000; i++) {
        result = integrator.integrate(thrust_func, -1.3, 3.3);
    }
    std::cout << result.first << " " << result.second << std::endl;
}


void integrate()
{
    std::cout << "Integration test" << std::endl;

    test_integration<readdy::util::integration::integrator>();
    test_integration<readdy::util::thrust_integration::integrator<double>>();
    test_integration<readdy::util::thrust_finite_integration::integrator>();
}