#include "hip/hip_runtime.h"
#include "../../../include/lattice_model_impl/thrust/thrust_complex_gaussian_distribution.hpp"

#ifdef THRUST

// https://docs.nvidia.com/cuda/hiprand/device-api-overview.html#thrust-and-hiprand-example

struct perform_gaussian_complex_langevin_update
{
    const double a_real;
    const double a_imag;
    const uint autocorrelation;
    const double epsilon;
    const double sqrt2epsilon;
    const unsigned long long offset;

    perform_gaussian_complex_langevin_update(const double a_real, const double a_imag, const uint autocorrelation, const double epsilon, const unsigned long long offset) :
            a_real(a_real), a_imag(a_imag), autocorrelation(autocorrelation), epsilon(epsilon), sqrt2epsilon(std::sqrt(2.0 * epsilon)), offset(offset)
    {
        std::cout << "Epsilon: " << epsilon << std::endl;
        std::cout << "Sqrt2Epsilon: " << sqrt2epsilon << std::endl;
    }

    template <typename Tuple>
    __device__
    void operator()(Tuple t) {
        unsigned int seed = thrust::get<0>(t);

        hiprandState s;

        // seed a random number generator
        hiprand_init(seed, 0, offset, &s);

        for (unsigned int i = 0; i < autocorrelation; ++i) {
            double x = thrust::get<1>(t);
            thrust::get<1>(t) = thrust::get<1>(t) - epsilon * (a_real * thrust::get<1>(t) - a_imag * thrust::get<2>(t)) + sqrt2epsilon * hiprand_normal(&s); // normal(rng);
            thrust::get<2>(t) = thrust::get<2>(t) - epsilon * (a_real * thrust::get<2>(t) + a_imag * x);
        }
    }
};

struct second_moment_function : public thrust::binary_function<cudaT, cudaT, float>
{
    __device__
    thrust::complex<double> operator() (cudaT &real_val, cudaT &imag_val)
    {
        return {real_val * real_val - imag_val * imag_val, 2.0 * real_val * imag_val};
    }
};

struct fourth_moment_function : public thrust::binary_function<cudaT, cudaT, float>
{
    __device__
    thrust::complex<double> operator() (cudaT &real_val, cudaT &imag_val)
    {
        return {real_val * real_val * real_val * real_val - 6.0 * real_val * real_val * imag_val * imag_val + imag_val * imag_val * imag_val * imag_val,
                4.0 * real_val * imag_val * (real_val * real_val - imag_val * imag_val)};
    }
};

struct print_functor
{
    explicit print_functor(std::ofstream & os_) : os(os_)
    {}

    template <typename Tuple>
    __host__
    void operator() (Tuple t)
    {
        os << thrust::get<0>(t) << "\t" << thrust::get<1>(t) << std::endl;
    }

    std::ofstream & os;
};

thrust_complex_gaussian_distribution::thrust_complex_gaussian_distribution(std::complex<double> a_, uint n_autocorrelation_,
                                         uint n_initialization, const double epsilon_, const int M_, const std::string files_dir_) :
            a(a_), epsilon(epsilon_), n_autocorrelation(n_autocorrelation_), M(M_), total_updates(0), files_dir(files_dir_)
    {
        std::cout << "a_real: " << a.real() << "\ta_imag: " << a.imag() << std::endl;
        // allocate storage
        real_random_numbers = dev_vec (M, 0);
        imag_random_numbers = dev_vec (M, 0);
        // indices = dev_vec_int (M, 0);
        indices = thrust::host_vector<double> (M, 0);
        host_real_random_numbers = thrust::host_vector<int> (M, 0);
        host_imag_random_numbers = thrust::host_vector<int> (M, 0);

        dist = thrust::uniform_real_distribution<float>(0.0,1.0);

        if(files_dir != "None") {
            std::string rel_data_path =
                    "/data/" + files_dir + "_" + std::to_string(a.real()) + "_" + std::to_string(a.imag()) + "/";
            std::string filename = "expectation_value";
            if(boost::filesystem::is_directory(gcp() + rel_data_path)) {
                std::cout << "Clear data directory" << std::endl;
                boost::filesystem::path path_to_remove(gcp() + rel_data_path);
                for (boost::filesystem::directory_iterator end_dir_it, it(path_to_remove); it != end_dir_it; ++it) {
                    boost::filesystem::remove_all(it->path());
                }
            }
            std::cout << "Create and prepare directory new for storage of random numbers" << std::endl;
            boost::filesystem::create_directories(gcp() + rel_data_path);
            Fileos fileos (gcp() + rel_data_path + "/" +  filename + ".dat");
            auto& os = fileos.get();
            os << "StateReal\tStateImag" << std::endl;
        }

        update_random_numbers(n_initialization);
    }

void thrust_complex_gaussian_distribution::write_data_to_file(const std::string files_dir) const
{
    std::string rel_data_path = "/data/" + files_dir + "/";

    std::string filename = "expectation_value";
    std::cout << gcp() + rel_data_path + "/" +  filename + ".dat" << std::endl;
    Fileos fileos (gcp() + rel_data_path + "/" +  filename + ".dat", true);
    auto& os = fileos.get();
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(host_real_random_numbers.begin(), host_imag_random_numbers.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(host_real_random_numbers.end(), host_imag_random_numbers.end())),
                     print_functor(os));

}

void thrust_complex_gaussian_distribution::update_random_numbers(const uint n_updates)
{
    if(n_updates > 1000000)
    {
        std::cout << "Start evolution (total updates - " << n_updates << ")" << std::endl;
        auto c = 0;
        while(c < n_updates)
        {
            thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(0),
                                                                          real_random_numbers.begin(),
                                                                          imag_random_numbers.begin())),
                             thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(M),
                                                                          real_random_numbers.end(),
                                                                          imag_random_numbers.end())),
                             perform_gaussian_complex_langevin_update(a.real(), a.imag(), 1000000, epsilon, total_updates));
            std::cout << "At " << 100.0 * c / n_updates << " % " << std::endl;
            total_updates += 1000000;
            c += 1000000;
        }

    }
    else
    {
        std::cout << "Start evolution" << std::endl;
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(0),
                                                                  real_random_numbers.begin(),
                                                                  imag_random_numbers.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(M),
                                                         real_random_numbers.end(),
                                                         imag_random_numbers.end())),
            perform_gaussian_complex_langevin_update(a.real(), a.imag(), n_updates, epsilon, total_updates));

        total_updates += n_updates;

    }


    std::cout << "Finished evolution, starting to sort" << std::endl;

    // Compute second moment
    thrust::device_vector<thrust::complex<double>> second_moment (M, 0);
    thrust::transform(real_random_numbers.begin(), real_random_numbers.end(), imag_random_numbers.begin(), second_moment.begin(), second_moment_function());
    thrust::complex<double> result = thrust::reduce(second_moment.begin(), second_moment.end());
    result = result / M;
    std::cout << "Second moment = " << result.real() << " + i " << result.imag() << std::endl;

    // Compute fourth moment
    thrust::device_vector<thrust::complex<double>> fourth_moment (M, 0);
    thrust::transform(real_random_numbers.begin(), real_random_numbers.end(), imag_random_numbers.begin(), fourth_moment.begin(), fourth_moment_function());
    thrust::complex<double> result_fourth_momentum = thrust::reduce(fourth_moment.begin(), fourth_moment.end());
    result_fourth_momentum = result_fourth_momentum / M;
    std::cout << "Fourth moment = " << result_fourth_momentum.real() << " + i " << result_fourth_momentum.imag() << std::endl;

    // Copy data to host vectors
    thrust::copy(real_random_numbers.begin(), real_random_numbers.end(), host_real_random_numbers.begin());
    thrust::copy(imag_random_numbers.begin(), imag_random_numbers.end(), host_imag_random_numbers.begin());

    // Write data to file
    if(files_dir != "None")
        write_data_to_file(files_dir + "_" + std::to_string(a.real()) + "_" + std::to_string(a.imag()));

    // Prepare permutation by assigning random uniform values to permutation_keys
    thrust::host_vector<double> permutation_keys(M, 0);
    thrust::sequence(indices.begin(), indices.end());
    thrust::generate(thrust::host, permutation_keys.begin(), permutation_keys.end(), [this] () { return dist(rng); });

    // Sort indices to obtain a permutation in indices
    thrust::host_vector<double> permutation_keys_dev(permutation_keys);
    thrust::sort_by_key(permutation_keys_dev.begin(), permutation_keys_dev.end(), indices.begin());

    // Generate iterator over random_numbers based on permutation in indices
    iter_real_random_numbers = thrust::permutation_iterator<thrust::host_vector<cudaT>::iterator, thrust::host_vector<int>::iterator> (
            host_real_random_numbers.begin(), indices.begin());
    iter_imag_random_numbers = thrust::permutation_iterator<thrust::host_vector<cudaT>::iterator, thrust::host_vector<int>::iterator> (
            host_imag_random_numbers.begin(), indices.begin());

    i = 0;

    /* print_range("Random Numbers", real_random_numbers.begin(), real_random_numbers.end());
    print_range("Permutation", indices.begin(), indices.end());
    print_range("Iter", iter_random_numbers, iter_random_numbers + 10); */
}

double thrust_complex_gaussian_distribution::get_random_number()
{
    if(i == M - 1) {
        update_random_numbers(n_autocorrelation);
        return iter_real_random_numbers[i];
    }
    else {
        i += 1;
        return iter_real_random_numbers[i];
    }
}

std::complex<double> thrust_complex_gaussian_distribution::get_complex_random_number()
{
    if(i == M - 1) {
        update_random_numbers(n_autocorrelation);
        return std::complex<double> {iter_real_random_numbers[i], iter_imag_random_numbers[i]};
    }
    else {
        i += 1;
        return std::complex<double> {iter_real_random_numbers[i], iter_imag_random_numbers[i]};
    }
}

#endif