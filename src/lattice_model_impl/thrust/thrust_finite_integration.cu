#include "hip/hip_runtime.h"
#include "../../../include/lattice_model_impl/thrust/thrust_finite_integration.hpp"

#ifdef THRUST

template<typename Func>
struct triangle
{
    Func f;
    const double lower_limit;
    const double interval;

    triangle(Func &f_, const double lower_limit_, const double interval_) : f(f_), lower_limit(lower_limit_), interval(interval_)
    {}

    __host__ __device__
    double operator() (const int& n)
    {
        return 0.5 * interval * (f(lower_limit + n * interval) + f(lower_limit + (n + 1) * interval));
    }
};


template<typename Func>
struct square
{
    Func f;
    const double lower_limit;
    const double interval;

    square(Func &f_, const double lower_limit_, const double interval_) : f(f_), lower_limit(lower_limit_), interval(interval_)
    {}

    __host__ __device__
    double operator() (const int& n)
    {
        return f(lower_limit + n * interval) * interval;
    }
};


inline readdy::util::thrust_finite_integration::integrator::integrator(const int n_) :
    n(n_)
{
    // results_sq.resize(n);
    results.resize(n);
}


template<typename Func, typename ScalarType>
inline std::pair<ScalarType, ScalarType> readdy::util::thrust_finite_integration::integrator::integrate(Func f, ScalarType lowerLimit, ScalarType upperLimit, bool watch)
{
    // std::cout << lowerLimit << " " << upperLimit << std::endl;

    const double interval = (upperLimit - lowerLimit) / n;

    // thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator(n), results_sq.begin(), square<Func>(f, lowerLimit, interval));
    thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator(n), results.begin(), triangle<Func>(f, lowerLimit, interval));

    // ScalarType integral_sq = thrust::reduce(results_sq.begin(), results_sq.end(), 0.0, thrust::plus<ScalarType>());
    ScalarType integral = thrust::reduce(results.begin(), results.end(), 0.0, thrust::plus<ScalarType>());

    // ScalarType integral_sq = thrust::transform_reduce(thrust::make_counting_iterator(0), thrust::make_counting_iterator(n), square<Func>(f, lowerLimit, interval), 0.0, thrust::plus<ScalarType>());
    // ScalarType integral = thrust::transform_reduce(thrust::make_counting_iterator(0), thrust::make_counting_iterator(n), triangle<Func>(f, lowerLimit, interval), 0.0, thrust::plus<ScalarType>());

    // std::cout << "Integrals" << integral << "\t" << integral_sq << std::endl;

    return std::make_pair(integral, 0.0); // abs(integral - integral_sq));
}
#endif