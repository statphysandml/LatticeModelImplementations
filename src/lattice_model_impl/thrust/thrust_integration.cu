#include "hip/hip_runtime.h"
#include "../../../include/lattice_model_impl/thrust/thrust_integration.hpp"

#ifdef THRUST

template<typename Func>
struct transfo
{
    Func f;
    const double midpoint;
    const double halfInterval;

    transfo(Func f_, const double midpoint_, const double halfInterval_) : f(f_), midpoint(midpoint_), halfInterval(halfInterval_)
    {}

    __host__ __device__
    double operator() (const double& abscissa, const double& weight)
    {
        return halfInterval * f(halfInterval * abscissa + midpoint) * weight;
    }
};

template<typename ScalarType>
struct negate
{
    __host__ __device__
    ScalarType operator() (const ScalarType& val)
    {
        return -1.0 * val;
    }
};

template<typename scalar_type>
inline readdy::util::thrust_integration::integrator<scalar_type>::integrator(const int) :
    len(abscissaeGaussKronrod201<scalar_type>.size()),lenGauss(abscissaeGauss201<scalar_type>.size())
{
    thrustweightsGaussKronrod201.resize(2 * len - 1);
    thrustabscissaeGaussKronrod201.resize(2 * len - 1);
    thrustweightsGauss201.resize(2 * lenGauss);

    resultsGauss.resize(2 * lenGauss);
    results.resize(2 * len - 1);
    abscissae.resize(2 * len - 1);
    indices.resize(2 * len - 1);
    thrust::sequence(indices.begin(), indices.end());

    thrust::copy(weightsGauss201<scalar_type>.begin(), weightsGauss201<scalar_type>.end(), thrustweightsGauss201.begin());
    thrust::copy(weightsGauss201<scalar_type>.rbegin(), weightsGauss201<scalar_type>.rend(), thrustweightsGauss201.begin() + lenGauss);

    thrust::copy(weightsGaussKronrod201<scalar_type>.begin(), weightsGaussKronrod201<scalar_type>.end(), thrustweightsGaussKronrod201.begin());
    thrust::copy(weightsGaussKronrod201<scalar_type>.rbegin() + 1, weightsGaussKronrod201<scalar_type>.rend(), thrustweightsGaussKronrod201.begin() + len);

    thrust::copy(abscissaeGaussKronrod201<scalar_type>.begin(), abscissaeGaussKronrod201<scalar_type>.end(), thrustabscissaeGaussKronrod201.begin());

    // Set weights
    thrust::transform(thrustabscissaeGaussKronrod201.rbegin() + len, thrustabscissaeGaussKronrod201.rend(), thrustabscissaeGaussKronrod201.begin() + len, negate<scalar_type>());
}

template<typename scalar_type>
template<typename Func, typename ScalarType>
inline std::pair<ScalarType, ScalarType> readdy::util::thrust_integration::integrator<scalar_type>::integrate(Func f, ScalarType lowerLimit, ScalarType upperLimit, bool watch)
{
    if (lowerLimit > upperLimit) {
        throw std::invalid_argument("lower limit cannot be larger than upper limit");
    } else if (lowerLimit == upperLimit) {
        return std::make_pair(static_cast<ScalarType>(0.), static_cast<ScalarType>(0.));
    }

    const ScalarType midpoint = (lowerLimit + upperLimit) / 2.;
    const ScalarType halfInterval = (upperLimit - lowerLimit) / 2.;

/*    thrust::transform(iw.thrustabscissaeGaussKronrod201.begin(), iw.thrustabscissaeGaussKronrod201.end(), iw.abscissae.begin(),
            [halfInterval, midpoint] __host__ __device__ (const ScalarType &abscissa)
            {
                return halfInterval * abscissa + midpoint;
            });

    // thrust::transform(iw.abscissae.begin(), iw.abscissae.end(), iw.abscissae.begin(), f);*/
    thrust::transform(thrustabscissaeGaussKronrod201.begin(), thrustabscissaeGaussKronrod201.end(), thrustweightsGaussKronrod201.begin(), results.begin(), transfo<Func>(f, midpoint, halfInterval));
    /* thrust::transform(iw.abscissae.begin(), iw.abscissae.end(), iw.thrustweightsGaussKronrod201.begin(), iw.results.begin(),
                      [halfInterval] __host__ __device__ (const ScalarType& val, const ScalarType& weight) { return halfInterval * val * weight; }); */
    ScalarType integralKronrod = thrust::reduce(results.begin(), results.end(), 0.0, thrust::plus<ScalarType>());

    /* auto values_end = thrust::remove_copy_if(iw.abscissae.begin(), iw.abscissae.end(), iw.indices.begin(), iw.abscissae.begin(), is_even<int>());
    iw.abscissae.resize(values_end - iw.abscissae.begin());

    thrust::transform(iw.abscissae.begin(), iw.abscissae.end(), iw.thrustweightsGauss201.begin(), iw.resultsGauss.begin(),
                      [halfInterval] __host__ __device__ (const ScalarType& val, const ScalarType& weight) { return halfInterval * val * weight; });
    ScalarType integralGauss = thrust::reduce(iw.resultsGauss.begin(), iw.resultsGauss.end(), 0.0, thrust::plus<ScalarType>()); */

    const ScalarType absoluteErrorEstimate = std::abs(integralKronrod - 1.0);
    return std::make_pair(integralKronrod, 0.0); // absoluteErrorEstimate);
}

// template class readdy::util::thrust_integration::IntegrationWeights<double>;

#endif